
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/types.h>
#include <semaphore.h>
#include <fcntl.h>
#include <sys/wait.h>
#include <unistd.h>
#include <hiprand/hiprand.h>
#define pi 3.141592653589
#define dabs(x) ((x) < 0 ? -(x) : (x))
#define UNSIGNED_INT_MAX 4294967296

// settings
double spaceBetweenRows = 10;
double needleSize = 7.853981633975;

int numberOfIterations = 280;
int CUDAthreads = 1024;
int CUDAblocks = 4096;

long numberOfThrowsPerIteration = CUDAthreads*CUDAblocks;
long numberOfThrows = numberOfThrowsPerIteration * numberOfIterations;;

__global__ void throwNeedles(hiprandGenerator_t gen, unsigned int *hitsArr, unsigned int *randArr, double spaceBetweenRows, double needleSize, long numberOfThrowsPerIteration, long numberOfThrows)
{
    double midPoint, heightDiff;
    long index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > numberOfThrowsPerIteration)
    {
        return;
    }
    
    double angle = randArr[index]; 
    midPoint = angle / (UNSIGNED_INT_MAX / spaceBetweenRows);
    heightDiff = needleSize / 2 * sin(angle);
    heightDiff = dabs(heightDiff);

    double height = dabs(spaceBetweenRows/2 - midPoint);
    if (heightDiff >= height)
    {
        hitsArr[index] += 1;
    }
}

int main(int argc, char *argv[])
{
    unsigned int *randArr, *hitsArr;
    time_t startTime, endTime;
    time(&startTime);

    hipMallocManaged(&randArr, sizeof(unsigned int)*numberOfThrowsPerIteration);
    hipMallocManaged(&hitsArr, sizeof(unsigned int)*numberOfThrowsPerIteration);
    hipMemset(&hitsArr, 0, numberOfThrowsPerIteration);
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)*clock());

    long hits = 0;

    for (int thread = 0; thread < numberOfIterations; thread++){
        hiprandGenerate(gen, randArr, numberOfThrowsPerIteration);

        throwNeedles<<<CUDAblocks, CUDAthreads>>>(gen,hitsArr, randArr, spaceBetweenRows, needleSize, numberOfThrowsPerIteration, numberOfIterations*numberOfThrowsPerIteration);
        
    }
    hipDeviceSynchronize();

    time(&endTime);
    double timeElapsed = difftime(endTime, startTime);

    for(long i = 0; i < numberOfThrowsPerIteration; i++)
    {
        hits += hitsArr[i];
    }

    time(&endTime);
    timeElapsed = difftime(endTime, startTime);

    double calculatedPi = 2 * needleSize * numberOfThrows / (hits * spaceBetweenRows);

    printf("Calculated pi: %lf\n", calculatedPi);
    printf("Actual pi:     %lf\n", pi);
    printf("Difference:    %lf\n", pi - calculatedPi);
    printf("Hit rate:      %lf\n", (double)hits / numberOfThrows);
    printf("Hits:          %ld\n", hits);
    printf("Expected hits: %ld\n", (long)(2 * needleSize * numberOfThrows / (pi * spaceBetweenRows)));
    printf("Program time:  %d sec\n", (int)timeElapsed);
    printf("\n");

    hipFree(randArr);
    hipFree(hitsArr);
    return 0;
}